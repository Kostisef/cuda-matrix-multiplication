#include "hip/hip_runtime.h"


#include <stdio.h>
//#include <conio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>


#define THREADSX 32
#define THREADSY 32


#define cudaCheckError() {										\
	hipError_t e = hipGetLastError();							\
	if (e != hipSuccess) {										\
		printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__,	\
		hipGetErrorString(e));									\
		exit(1);												\
	}															\
}


__global__ void MatrixMulKernel(double *A_d, double *C_d, int m, int n) {

	int row = blockIdx.y * blockDim.y +  threadIdx.y;
	int col = blockIdx.x * blockDim.x +  threadIdx.x;
	int k;

	if ((row < n) && (col < n))
	{
		double sum = 0;
		for (k = 0; k < m; k++)
		{
			sum += A_d[k*n + col] * A_d[k*n + row ];
		}
		C_d[row*n + col] = sum;
	}
}


void MatrixMultiplication(double *a, double *mul, int m, int n) {

	double BlockX;
	double BlockY;


//	int size = m * n * sizeof(double);
//	printf("Size of A:%d\n",size);
//	long int size2 = n * n * sizeof(double);

//	printf("Size of Product:%ld\n",size2);
	double *A_d, *C_d;


	hipMalloc((void**)&A_d, m * n * sizeof(double));
	cudaCheckError();

	hipMemcpy(A_d, a, m * n * sizeof(double), hipMemcpyHostToDevice);
	cudaCheckError();

	hipMalloc((void**)&C_d, n * n * sizeof(double));
	cudaCheckError();
	//dim3 grid(1, 1);

	//threadX = ceil(double(n / BLOCKX));
	//threadY = ceil(double(n / BLOCKY));

	BlockX = ceil((double)n / THREADSX);
	BlockY = ceil((double)n / THREADSY);


	if (THREADSX >= n)
	{
		BlockX = 1;
	}
	if (THREADSY >= n)
	{
		BlockY = 1;
	}

	dim3 grid(BlockX,BlockY);
	dim3 block(THREADSX, THREADSY);

	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	MatrixMulKernel <<< grid, block >>>(A_d, C_d, m, n);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("Time to generate :%f ms \n", time);

	cudaCheckError();

	hipMemcpy(mul, C_d, n * n * sizeof(double), hipMemcpyDeviceToHost);
	cudaCheckError();

	hipFree(A_d);
	cudaCheckError();
	hipFree(C_d);
	cudaCheckError();

}

void CpuMatrixMultiplication(double *A, double *C, int m, int n){
	double *mul_2;

	mul_2 = (double*)malloc(sizeof(double)*n*n);
	int i, j;
	double diff = 0;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			mul_2[i*n + j] = 0;
			for (int k = 0; k < m; k++)
			{
				mul_2[i*n + j] = mul_2[i*n + j] + A[n*k + i] * A[k*n + j];

			}
			// printf("A[%d][%d]--->%lf", i, j, A[i*n + j]);
			// printf("C[%d][%d]--->%lf\t", i, j,  C[i*n + j]);
		}
		// printf("\n");
	}

	/* CALCULATE THE DIFFERENCE BETWEEN THE MATRIX CALCULATED FROM GPU AND THE MATRIX CALCULATED IN CPU*/
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			if (diff != 0)
			{
				break;
			}
			else
			{
				diff += C[i*n + j] - mul_2[i*n + j];
			}
		}
		if (diff != 0)
		{
			break;
		}
	}

	printf("diff --> %lf\n", diff);
	free(mul_2);
}


int main(int argc,char* argv[]) {


	int m, n, i, j;

	//printf("Enter order of matrix A: ");
	//scanf_s("%d%d", &m, &n);

	if (argc == 3){
		m = atoi(argv[1]);
		n = atoi(argv[2]);
	}
	else{
			printf("Error: Invalid number of arguments");
			return 1;
	}

	double *A;
	double *C;
	time_t t;

	A = (double*)malloc(sizeof(double)*m*n);  // host memory for A
	C = (double*)malloc(sizeof(double)*n*n);  // host memory for C

	srand((unsigned)time(&t));

	for (i = 0; i < m; i++)
	{
		for (j = 0; j < n; j++)
		{
			// printf("Enter value of a[%d][%d]: ", i, j);
			// scanf("%lf", &A[n*i + j]);
			A[n*i + j] = ((double)rand() / (double)RAND_MAX);
			
			// printf("A[%d][%d]-->%.4lf \t", i, j, A[n*i + j]);
		}
		// printf("\n");
	}

	printf("\tStart1\n" );
	MatrixMultiplication(A,C, m, n);

	// CpuMatrixMultiplication(A, C, m, n);


	free(C);
	free(A);

	//_getch();
	return 0;


}




